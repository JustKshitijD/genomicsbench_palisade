#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "matcher_gpu.cuh"

#include <thrust/scan.h>
#include <thrust/transform_scan.h>
#include <thrust/execution_policy.h>
#include <cassert>

#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/utils/mathutils.hpp>
#include <claragenomics/utils/signed_integer_utils.hpp>

namespace
{
template <typename RandomAccessIterator, typename ValueType>
__device__ RandomAccessIterator lower_bound(RandomAccessIterator lower_bound, RandomAccessIterator upper_bound, ValueType query)
{
    assert(upper_bound >= lower_bound);
    while (upper_bound > lower_bound)
    {
        RandomAccessIterator mid = lower_bound + (upper_bound - lower_bound) / 2;
        const auto mid_value     = *mid;
        if (mid_value < query)
            lower_bound = mid + 1;
        else
            upper_bound = mid;
    }
    return lower_bound;
}

template <typename RandomAccessIterator, typename ValueType>
__device__ RandomAccessIterator upper_bound(RandomAccessIterator lower_bound, RandomAccessIterator upper_bound, ValueType query)
{
    assert(upper_bound >= lower_bound);
    while (upper_bound > lower_bound)
    {
        RandomAccessIterator mid = lower_bound + (upper_bound - lower_bound) / 2;
        const auto mid_value     = *mid;
        if (mid_value <= query)
            lower_bound = mid + 1;
        else
            upper_bound = mid;
    }
    return lower_bound;
}
} // namespace

namespace claragenomics
{

namespace cudamapper
{

MatcherGPU::MatcherGPU(const Index& query_index,
                       const Index& target_index)
{

    CGA_NVTX_RANGE(profile, "matcherGPU");
    if (query_index.number_of_reads() == 0 || target_index.number_of_reads() == 0)
        return;

    // We need to compute a set of anchors between the query and the target.
    // An anchor is a combination of a query (read_id, position) and
    // target {read_id, position} with the same representation.
    // The set of anchors of a matching query and target representation
    // is the all-to-all combination of the corresponding set of {(read_id, position)}
    // of the query with the set of {(read_id, position)} of the target.
    //
    // We compute the anchors for each unique representation of the query index.
    // The array index of the following data structures will correspond to the array index of the
    // unique representation in the query index.

    thrust::device_vector<std::int64_t> found_target_indices_d(query_index.unique_representations().size());
    thrust::device_vector<std::int64_t> anchor_starting_indices_d(query_index.unique_representations().size());

    // First we search for each unique representation of the query index, the array index
    // of the same representation in the array of unique representations of target index
    // (or -1 if representation is not found).
    details::matcher_gpu::find_query_target_matches(found_target_indices_d, query_index.unique_representations(), target_index.unique_representations());

    // For each unique representation of the query index compute the number of corrsponding anchors
    // and store the resulting starting index in an anchors array if all anchors are stored in a flat array.
    // The last element will be the total number of anchors.
    details::matcher_gpu::compute_anchor_starting_indices(anchor_starting_indices_d, query_index.first_occurrence_of_representations(), found_target_indices_d, target_index.first_occurrence_of_representations());

    const int64_t n_anchors = anchor_starting_indices_d.back(); // D->H transfer

    anchors_d_.resize(n_anchors);

    // Generate the anchors
    // by computing the all-to-all combinations of the matching representations in query and target
    details::matcher_gpu::generate_anchors(anchors_d_,
                                           anchor_starting_indices_d,
                                           query_index.first_occurrence_of_representations(),
                                           found_target_indices_d,
                                           target_index.first_occurrence_of_representations(),
                                           query_index.read_ids(),
                                           query_index.positions_in_reads(),
                                           target_index.read_ids(),
                                           target_index.positions_in_reads());
}

thrust::device_vector<Anchor>& MatcherGPU::anchors()
{
    return anchors_d_;
}

namespace details
{

namespace matcher_gpu
{

void find_query_target_matches(
    thrust::device_vector<std::int64_t>& found_target_indices_d,
    const thrust::device_vector<representation_t>& query_representations_d,
    const thrust::device_vector<representation_t>& target_representations_d)
{
    assert(found_target_indices_d.size() == query_representations_d.size());

    const int32_t n_threads = 256;
    const int32_t n_blocks  = ceiling_divide<int64_t>(query_representations_d.size(), n_threads);

    find_query_target_matches_kernel<<<n_blocks, n_threads>>>(found_target_indices_d.data().get(), query_representations_d.data().get(), get_size(query_representations_d), target_representations_d.data().get(), get_size(target_representations_d));
}

void compute_anchor_starting_indices(
    thrust::device_vector<std::int64_t>& anchor_starting_indices_d,
    const thrust::device_vector<std::uint32_t>& query_starting_index_of_each_representation_d,
    const thrust::device_vector<std::int64_t>& found_target_indices_d,
    const thrust::device_vector<std::uint32_t>& target_starting_index_of_each_representation_d)
{
    assert(query_starting_index_of_each_representation_d.size() == found_target_indices_d.size() + 1);
    assert(anchor_starting_indices_d.size() == found_target_indices_d.size());

    const std::uint32_t* const query_starting_indices  = query_starting_index_of_each_representation_d.data().get();
    const std::uint32_t* const target_starting_indices = target_starting_index_of_each_representation_d.data().get();
    const std::int64_t* const found_target_indices     = found_target_indices_d.data().get();

    thrust::transform_inclusive_scan(
        thrust::make_counting_iterator(std::int64_t(0)),
        thrust::make_counting_iterator(get_size(anchor_starting_indices_d)),
        anchor_starting_indices_d.begin(),
        [query_starting_indices, target_starting_indices, found_target_indices] __device__(std::uint32_t query_index) -> std::int64_t {
            std::int32_t n_queries_with_representation = query_starting_indices[query_index + 1] - query_starting_indices[query_index];
            std::int64_t target_index                  = found_target_indices[query_index];
            std::int32_t n_targets_with_representation = 0;
            if (target_index >= 0)
                n_targets_with_representation = target_starting_indices[target_index + 1] - target_starting_indices[target_index];
            return n_queries_with_representation * n_targets_with_representation;
        },
        thrust::plus<std::int64_t>());
}

__global__ void find_query_target_matches_kernel(
    int64_t* const found_target_indices,
    const representation_t* const query_representations_d,
    const int64_t n_query_representations,
    const representation_t* const target_representations_d,
    const int64_t n_target_representations)
{
    const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_query_representations)
        return;

    const representation_t query = query_representations_d[i];
    int64_t found_target_index   = -1;
    const representation_t* lb   = lower_bound(target_representations_d, target_representations_d + n_target_representations, query);
    if (*lb == query)
        found_target_index = lb - target_representations_d;

    found_target_indices[i] = found_target_index;
}

void generate_anchors(
    thrust::device_vector<Anchor>& anchors,
    const thrust::device_vector<std::int64_t>& anchor_starting_indices_d,
    const thrust::device_vector<std::uint32_t>& query_starting_index_of_each_representation_d,
    const thrust::device_vector<std::int64_t>& found_target_indices_d,
    const thrust::device_vector<std::uint32_t>& target_starting_index_of_each_representation_d,
    const thrust::device_vector<read_id_t>& query_read_ids,
    const thrust::device_vector<position_in_read_t>& query_positions_in_read,
    const thrust::device_vector<read_id_t>& target_read_ids,
    const thrust::device_vector<position_in_read_t>& target_positions_in_read)
{
    assert(anchor_starting_indices_d.size() + 1 == query_starting_index_of_each_representation_d.size());
    assert(found_target_indices_d.size() + 1 == query_starting_index_of_each_representation_d.size());
    assert(query_read_ids.size() == query_positions_in_read.size());
    assert(target_read_ids.size() == target_positions_in_read.size());

    const int32_t n_threads = 256;
    const int32_t n_blocks  = ceiling_divide<int64_t>(get_size(anchors), n_threads);
    generate_anchors_kernel<<<n_blocks, n_threads>>>(
        anchors.data().get(),
        get_size(anchors),
        anchor_starting_indices_d.data().get(),
        query_starting_index_of_each_representation_d.data().get(),
        found_target_indices_d.data().get(),
        get_size(found_target_indices_d),
        target_starting_index_of_each_representation_d.data().get(),
        query_read_ids.data().get(),
        query_positions_in_read.data().get(),
        target_read_ids.data().get(),
        target_positions_in_read.data().get());
}

__global__ void generate_anchors_kernel(
    Anchor* const anchors_d,
    const int64_t n_anchors,
    const int64_t* const anchor_starting_index_d,
    const std::uint32_t* const query_starting_index_of_each_representation_d,
    const std::int64_t* const found_target_indices_d,
    const int32_t n_query_representations,
    const std::uint32_t* const target_starting_index_of_each_representation_d,
    const read_id_t* const query_read_ids,
    const position_in_read_t* const query_positions_in_read,
    const read_id_t* const target_read_ids,
    const position_in_read_t* const target_positions_in_read)
{
    // Fill the anchor_d array. Each thread generates one anchor.
    std::int64_t anchor_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (anchor_idx >= n_anchors)
        return;

    // Figure out for which representation this thread should compute the anchor.
    // We only need the index in the unique representation array of the query index
    // not the representation itself.
    const std::int64_t representation_idx = upper_bound(anchor_starting_index_d, anchor_starting_index_d + n_query_representations, anchor_idx) - anchor_starting_index_d;

    assert(representation_idx < n_query_representations);

    // Compute the index of the anchor within only this representation.
    std::uint32_t relative_anchor_index = anchor_idx;
    if (representation_idx > 0)
        relative_anchor_index -= anchor_starting_index_d[representation_idx - 1];

    // Get the ranges within the query and target index with this representation.
    const std::int64_t j = found_target_indices_d[representation_idx];
    assert(j >= 0);
    const std::uint32_t query_begin  = query_starting_index_of_each_representation_d[representation_idx];
    const std::uint32_t target_begin = target_starting_index_of_each_representation_d[j];
    const std::uint32_t target_end   = target_starting_index_of_each_representation_d[j + 1];

    const std::uint32_t n_targets = target_end - target_begin;

    // Overall we want to do an all-to-all (n*m) matching between the query and target entries
    // with the same representation.
    // Compute the exact combination query and target index entry for which
    // we generate the anchor in this thread.
    const std::uint32_t query_idx  = query_begin + relative_anchor_index / n_targets;
    const std::uint32_t target_idx = target_begin + relative_anchor_index % n_targets;

    assert(query_idx < query_starting_index_of_each_representation_d[representation_idx + 1]);

    // Generate and store the anchor
    Anchor a;
    a.query_read_id_           = query_read_ids[query_idx];
    a.target_read_id_          = target_read_ids[target_idx];
    a.query_position_in_read_  = query_positions_in_read[query_idx];
    a.target_position_in_read_ = target_positions_in_read[target_idx];
    anchors_d[anchor_idx]      = a;
}

} // namespace matcher_gpu

} // namespace details
} // namespace cudamapper

} // namespace claragenomics
