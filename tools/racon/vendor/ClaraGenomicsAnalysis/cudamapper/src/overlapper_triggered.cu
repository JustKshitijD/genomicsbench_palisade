/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <claragenomics/utils/cudautils.hpp>
#include "cudamapper_utils.hpp"
#include "overlapper_triggered.hpp"
#include <fstream>

namespace claragenomics
{
namespace cudamapper
{

__host__ __device__ bool operator==(const Anchor& lhs,
                                    const Anchor& rhs)
{
    auto score_threshold = 1;

    // Very simple scoring function to quantify quality of overlaps.
    // TODO change to a more sophisticated scoring method
    auto score = 1;
    if ((rhs.query_position_in_read_ - lhs.query_position_in_read_) < 350)
        score = 2;

    return ((lhs.query_read_id_ == rhs.query_read_id_) &&
            (lhs.target_read_id_ == rhs.target_read_id_) &&
            score > score_threshold);
}

struct cuOverlapKey
{
    const Anchor* anchor;
};

struct cuOverlapKey_transform
{
    const Anchor* d_anchors;
    const int32_t* d_chain_start;

    cuOverlapKey_transform(const Anchor* anchors, const int32_t* chain_start)
        : d_anchors(anchors)
        , d_chain_start(chain_start)
    {
    }

    __host__ __device__ __forceinline__ cuOverlapKey
    operator()(const int32_t& idx) const
    {
        auto anchor_idx = d_chain_start[idx];

        cuOverlapKey key;
        key.anchor = &d_anchors[anchor_idx];
        return key;
    }
};

__host__ __device__ bool operator==(const cuOverlapKey& key0,
                                    const cuOverlapKey& key1)
{
    const Anchor* a = key0.anchor;
    const Anchor* b = key1.anchor;
    return (a->target_read_id_ == b->target_read_id_) &&
           (a->query_read_id_ == b->query_read_id_);
}

struct cuOverlapArgs
{
    int32_t overlap_end;
    int32_t num_residues;
    int32_t overlap_start;
};

struct cuOverlapArgs_transform
{
    const int32_t* d_chain_start;
    const int32_t* d_chain_length;

    cuOverlapArgs_transform(const int32_t* chain_start, const int32_t* chain_length)
        : d_chain_start(chain_start)
        , d_chain_length(chain_length)
    {
    }

    __host__ __device__ __forceinline__ cuOverlapArgs
    operator()(const int32_t& idx) const
    {
        cuOverlapArgs overlap;
        auto overlap_start    = d_chain_start[idx];
        auto overlap_length   = d_chain_length[idx];
        overlap.overlap_end   = overlap_start + overlap_length;
        overlap.num_residues  = overlap_length;
        overlap.overlap_start = overlap_start;
        return overlap;
    }
};

struct FuseOverlapOp
{
    __host__ __device__ cuOverlapArgs operator()(const cuOverlapArgs& a,
                                                 const cuOverlapArgs& b) const
    {
        cuOverlapArgs fused_overlap;
        fused_overlap.num_residues = a.num_residues + b.num_residues;
        fused_overlap.overlap_end =
            a.overlap_end > b.overlap_end ? a.overlap_end : b.overlap_end;
        fused_overlap.overlap_start =
            a.overlap_start < b.overlap_start ? a.overlap_start : b.overlap_start;
        return fused_overlap;
    }
};

struct CreateOverlap
{
    const Anchor* d_anchors;

    __host__ __device__ __forceinline__ CreateOverlap(const Anchor* anchors_ptr)
        : d_anchors(anchors_ptr)
    {
    }

    __host__ __device__ __forceinline__ Overlap
    operator()(cuOverlapArgs overlap)
    {
        Anchor overlap_start_anchor = d_anchors[overlap.overlap_start];
        Anchor overlap_end_anchor   = d_anchors[overlap.overlap_end - 1];

        Overlap new_overlap;

        new_overlap.query_read_id_  = overlap_end_anchor.query_read_id_;
        new_overlap.target_read_id_ = overlap_end_anchor.target_read_id_;
        new_overlap.num_residues_   = overlap.num_residues;
        new_overlap.target_end_position_in_read_ =
            overlap_end_anchor.target_position_in_read_;
        new_overlap.target_start_position_in_read_ =
            overlap_start_anchor.target_position_in_read_;
        new_overlap.query_end_position_in_read_ =
            overlap_end_anchor.query_position_in_read_;
        new_overlap.query_start_position_in_read_ =
            overlap_start_anchor.query_position_in_read_;
        new_overlap.overlap_complete = true;

        // If the target start position is greater than the target end position
        // We can safely assume that the query and target are template and
        // complement reads. TODO: Incorporate sketchelement direction value when
        // this is implemented
        if (new_overlap.target_start_position_in_read_ >
            new_overlap.target_end_position_in_read_)
        {
            new_overlap.relative_strand = RelativeStrand::Reverse;
            auto tmp                    = new_overlap.target_end_position_in_read_;
            new_overlap.target_end_position_in_read_ =
                new_overlap.target_start_position_in_read_;
            new_overlap.target_start_position_in_read_ = tmp;
        }
        else
        {
            new_overlap.relative_strand = RelativeStrand::Forward;
        }
        return new_overlap;
    };
};

void OverlapperTriggered::get_overlaps(std::vector<Overlap>& fused_overlaps,
                                       thrust::device_vector<Anchor>& d_anchors,
                                       const Index& index_query,
                                       const Index& index_target)
{
    CGA_NVTX_RANGE(profiler, "OverlapperTriggered::get_overlaps");
    const auto tail_length_for_chain = 3;
    auto n_anchors                   = d_anchors.size();

    // comparison operator - lambda used to compare Anchors in sort
    auto comp = [] __host__ __device__(const Anchor& i, const Anchor& j) -> bool {
        return (i.query_read_id_ < j.query_read_id_) ||
               ((i.query_read_id_ == j.query_read_id_) &&
                (i.target_read_id_ < j.target_read_id_)) ||
               ((i.query_read_id_ == j.query_read_id_) &&
                (i.target_read_id_ == j.target_read_id_) &&
                (i.query_position_in_read_ < j.query_position_in_read_)) ||
               ((i.query_read_id_ == j.query_read_id_) &&
                (i.target_read_id_ == j.target_read_id_) &&
                (i.query_position_in_read_ == j.query_position_in_read_) &&
                (i.target_position_in_read_ < j.target_position_in_read_));
    };

    // sort on device
    // TODO : currently thrust::sort requires O(2N) auxiliary storage, implement the same functionality using O(N) auxiliary storage
    thrust::sort(thrust::device, d_anchors.begin(), d_anchors.end(), comp);

    // temporary workspace buffer on device
    thrust::device_vector<char> d_temp_buf;

    // Do run length encode to compute the chains
    // note - identifies the start and end anchor of the chain without moving the anchors
    // >>>>>>>>>

    // d_start_anchor[i] contains the starting anchor of chain i
    thrust::device_vector<Anchor> d_start_anchor(n_anchors);

    // d_chain_length[i] contains the length of chain i
    thrust::device_vector<int32_t> d_chain_length(n_anchors);

    // total number of chains found
    thrust::device_vector<int32_t> d_nchains(1);

    void* d_temp_storage      = nullptr;
    size_t temp_storage_bytes = 0;
    // calculate storage requirement for run length encoding
    hipcub::DeviceRunLengthEncode::Encode(
        d_temp_storage, temp_storage_bytes, d_anchors.data(), d_start_anchor.data(),
        d_chain_length.data(), d_nchains.data(), n_anchors);

    // allocate temporary storage
    d_temp_buf.resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data().get();

    // run encoding
    hipcub::DeviceRunLengthEncode::Encode(
        d_temp_storage, temp_storage_bytes, d_anchors.data(), d_start_anchor.data(),
        d_chain_length.data(), d_nchains.data(), n_anchors);

    // <<<<<<<<<<

    // memcpy D2H
    auto n_chains = d_nchains[0];

    // use prefix sum to calculate the starting index position of all the chains
    // >>>>>>>>>>>>

    // for a chain i, d_chain_start[i] contains the index of starting anchor from d_anchors array
    thrust::device_vector<int32_t> d_chain_start(n_chains);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  d_chain_length.data(), d_chain_start.data(),
                                  n_chains);

    // allocate temporary storage
    d_temp_buf.resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data().get();

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  d_chain_length.data(), d_chain_start.data(),
                                  n_chains);

    // <<<<<<<<<<<<

    // calculate overlaps where overlap is a chain with length > tail_length_for_chain
    // >>>>>>>>>>>>

    // d_overlaps[j] contains index to d_chain_length/d_chain_start where
    // d_chain_length[d_overlaps[j]] and d_chain_start[d_overlaps[j]] corresponds
    // to length and index to starting anchor of the chain-d_overlaps[j] (also referred as overlap j)
    thrust::device_vector<int32_t> d_overlaps(n_chains);
    auto indices_end =
        thrust::copy_if(thrust::make_counting_iterator<int32_t>(0),
                        thrust::make_counting_iterator<int32_t>(n_chains),
                        d_chain_length.data(), d_overlaps.data(),
                        [=] __host__ __device__(const int32_t& len) -> bool {
                            return (len >= tail_length_for_chain);
                        });

    auto n_overlaps = indices_end - d_overlaps.data();
    // <<<<<<<<<<<<<

    // >>>>>>>>>>>>
    // fuse overlaps using reduce by key operations

    // key is a minimal data structure that is required to compare the overlaps
    cuOverlapKey_transform key_op(d_anchors.data().get(),
                                  d_chain_start.data().get());
    hipcub::TransformInputIterator<cuOverlapKey, cuOverlapKey_transform, int32_t*>
        d_keys_in(d_overlaps.data().get(),
                  key_op);

    // value is a minimal data structure that represents a overlap
    cuOverlapArgs_transform value_op(d_chain_start.data().get(),
                                     d_chain_length.data().get());

    hipcub::TransformInputIterator<cuOverlapArgs, cuOverlapArgs_transform, int32_t*>
        d_values_in(d_overlaps.data().get(),
                    value_op);

    thrust::device_vector<cuOverlapKey> d_fusedoverlap_keys(n_overlaps);
    thrust::device_vector<cuOverlapArgs> d_fusedoverlaps_args(n_overlaps);
    thrust::device_vector<int32_t> d_nfused_overlaps(1);

    FuseOverlapOp reduction_op;

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in,
                                   d_fusedoverlap_keys.data(), d_values_in,
                                   d_fusedoverlaps_args.data(), d_nfused_overlaps.data(),
                                   reduction_op, n_overlaps);

    // allocate temporary storage
    d_temp_buf.resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data().get();

    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in,
                                   d_fusedoverlap_keys.data(), d_values_in,
                                   d_fusedoverlaps_args.data(), d_nfused_overlaps.data(),
                                   reduction_op, n_overlaps);

    // memcpyD2H
    auto n_fused_overlap = d_nfused_overlaps[0];

    // construct overlap from the overlap args
    CreateOverlap fuse_op(d_anchors.data().get());
    thrust::device_vector<Overlap> d_fused_overlaps(n_fused_overlap);
    thrust::transform(d_fusedoverlaps_args.data(),
                      d_fusedoverlaps_args.data() + n_fused_overlap,
                      d_fused_overlaps.data(), fuse_op);

    // memcpyD2H - move fused overlaps to host
    fused_overlaps.resize(n_fused_overlap);
    thrust::copy(d_fused_overlaps.begin(), d_fused_overlaps.end(),
                 fused_overlaps.begin());
    // <<<<<<<<<<<<

    // parallel update the overlaps to include the corresponding read names [parallel on host]
    thrust::transform(thrust::host,
                      fused_overlaps.data(),
                      fused_overlaps.data() + n_fused_overlap,
                      fused_overlaps.data(), [&](Overlap& new_overlap) {
                          std::string query_read_name  = index_query.read_id_to_read_name(new_overlap.query_read_id_);
                          std::string target_read_name = index_target.read_id_to_read_name(new_overlap.target_read_id_);

                          new_overlap.query_read_name_ = new char[query_read_name.length()];
                          strcpy(new_overlap.query_read_name_, query_read_name.c_str());

                          new_overlap.target_read_name_ = new char[target_read_name.length()];
                          strcpy(new_overlap.target_read_name_, target_read_name.c_str());

                          new_overlap.query_length_  = index_query.read_id_to_read_length(new_overlap.query_read_id_);
                          new_overlap.target_length_ = index_target.read_id_to_read_length(new_overlap.target_read_id_);

                          return new_overlap;
                      });
}
} // namespace cudamapper
} // namespace claragenomics
