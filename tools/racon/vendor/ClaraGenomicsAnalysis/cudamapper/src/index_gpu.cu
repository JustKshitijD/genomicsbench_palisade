#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "index_gpu.cuh"

namespace claragenomics
{
namespace cudamapper
{
namespace details
{
namespace index_gpu
{
void find_first_occurrences_of_representations(thrust::device_vector<representation_t>& unique_representations_d,
                                               thrust::device_vector<std::uint32_t>& first_occurrence_index_d,
                                               const thrust::device_vector<representation_t>& input_representations_d)
{
    // each element has value 1 if representation with the same index in representations_d has a different value than it's neighbour to the left, 0 otehrwise
    // underlying type is 32-bit because a scan operation will be performed on the array, so the elements should be capable of holding a number that is equal to
    // the total number of 1s in the array
    thrust::device_vector<std::uint32_t> new_value_mask_d(input_representations_d.size());

    // TODO: Currently maximum number of thread blocks is 2^31-1. This means we support representations of up to (2^31-1) * number_of_threads
    // With 256 that's (2^31-1)*2^8 ~= 2^39. If representation is 4-byte (we expect it to be 4 or 8) that's 2^39*2^2 = 2^41 = 2TB. We don't expect to hit this limit any time soon
    // The kernel can be modified to process several representation per thread to support arbitrary size
    std::uint32_t number_of_threads = 256; // arbitrary value
    std::uint32_t number_of_blocks  = (input_representations_d.size() - 1) / number_of_threads + 1;

    create_new_value_mask<<<number_of_blocks, number_of_threads>>>(input_representations_d.data().get(),
                                                                   input_representations_d.size(),
                                                                   new_value_mask_d.data().get());

    // do inclusive scan
    // for example for
    // 0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20
    // 0  0  0  0 12 12 12 12 12 12 23 23 23 32 32 32 32 32 46 46 46
    // 1  0  0  0  1  0  0  0  0  0  1  0  0  1  0  0  0  0  1  0  0
    // gives
    // 1  1  1  1  2  2  2  2  2  2  3  3  3  4  4  4  4  4  5  5  5
    // meaning all elements with the same representation have the same value and those values are sorted in increasing order starting from 1
    thrust::device_vector<std::uint64_t> representation_index_mask_d(new_value_mask_d.size());
    thrust::inclusive_scan(thrust::device,
                           new_value_mask_d.begin(),
                           new_value_mask_d.end(),
                           representation_index_mask_d.begin());
    new_value_mask_d.clear();
    new_value_mask_d.shrink_to_fit();

    const std::uint64_t number_of_unique_representations = representation_index_mask_d.back(); // D2H copy

    first_occurrence_index_d.resize(number_of_unique_representations + 1); // <- +1 for the additional element
    first_occurrence_index_d.shrink_to_fit();
    unique_representations_d.resize(number_of_unique_representations);
    unique_representations_d.shrink_to_fit();

    find_first_occurrences_of_representations_kernel<<<number_of_blocks, number_of_threads>>>(representation_index_mask_d.data().get(),
                                                                                              input_representations_d.data().get(),
                                                                                              representation_index_mask_d.size(),
                                                                                              first_occurrence_index_d.data().get(),
                                                                                              unique_representations_d.data().get());
    // last element is the total number of elements in representations array
    first_occurrence_index_d.back() = input_representations_d.size(); // H2D copy
}

__global__ void create_new_value_mask(const representation_t* const representations_d,
                                      const std::size_t number_of_elements,
                                      std::uint32_t* const new_value_mask_d)
{
    std::uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= number_of_elements)
        return;

    if (index == 0)
    {
        new_value_mask_d[0] = 1;
    }
    else
    {
        if (representations_d[index] == representations_d[index - 1])
        {
            new_value_mask_d[index] = 0;
        }
        else
            new_value_mask_d[index] = 1;
    }
}

__global__ void find_first_occurrences_of_representations_kernel(const std::uint64_t* const representation_index_mask_d,
                                                                 const representation_t* const input_representations_d,
                                                                 const std::size_t number_of_input_elements,
                                                                 std::uint32_t* const starting_index_of_each_representation_d,
                                                                 representation_t* const unique_representations_d)
{
    // one thread per element of input_representations_d (i.e. sketch_element)
    std::uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= number_of_input_elements)
        return;

    if (index == 0)
    {
        starting_index_of_each_representation_d[0] = 0;
        unique_representations_d[0]                = input_representations_d[0];
    }
    else
    {
        // representation_index_mask_d gives a unique index to each representation, starting from 1, thus '-1'
        const auto representation_index_mask_for_this_index = representation_index_mask_d[index];
        if (representation_index_mask_for_this_index != representation_index_mask_d[index - 1])
        {
            // if new representation is not the same as its left neighbor
            // save the index at which that representation starts
            starting_index_of_each_representation_d[representation_index_mask_for_this_index - 1] = index;
            unique_representations_d[representation_index_mask_for_this_index - 1]                = input_representations_d[index];
        }
    }
}
} // namespace index_gpu
} // namespace details

} // namespace cudamapper
} // namespace claragenomics
