#include "hip/hip_runtime.h"
#include "common.h"
#include "datatypes.h"

__device__
score_dt device_ilog2(score_dt v)
{
    if (v < 2) return 0;
    else if (v < 4) return 1;
    else if (v < 8) return 2;
    else if (v < 16) return 3;
    else if (v < 32) return 4;
    else if (v < 64) return 5;
    else if (v < 128) return 6;
    else if (v < 256) return 7;
    else return 8;
}

__device__
score_dt chain_dp_score(anchor_dt *active, anchor_dt curr,
        float avg_qspan, int max_dist_x, int max_dist_y, int bw, int id)
{
    anchor_dt act;
    *((short4*)&act) = ((short4*)active)[id];

    if (curr.tag != act.tag) return NEG_INF_SCORE_GPU;

    score_dt dist_x = act.x - curr.x;
    if (dist_x == 0 || dist_x > max_dist_x) return NEG_INF_SCORE_GPU;

    score_dt dist_y = act.y - curr.y;
    if (dist_y > max_dist_y || dist_y <= 0) return NEG_INF_SCORE_GPU;

    score_dt dd = dist_x > dist_y ? dist_x - dist_y : dist_y - dist_x;
    if (dd > bw) return NEG_INF_SCORE_GPU;

    score_dt min_d = dist_y < dist_x ? dist_y : dist_x;
    score_dt log_dd = device_ilog2(dd);

    score_dt sc = min_d > act.w ? act.w : min_d;
    sc -= (score_dt)(dd * (0.01 * avg_qspan)) + (log_dd >> 1);

    return sc;
}

//#define USE_LOCAL_BUFFER

__global__
void device_chain_tiled(
        return_dt *ret, int n, const anchor_dt *a,
        control_dt *cont, score_dt **max_tracker_g, parent_dt **j_tracker_g,
        int max_dist_x, int max_dist_y, int bw)
{
    int block = blockIdx.x;
    int id = threadIdx.x % BACK_SEARCH_COUNT_GPU;
    int sub = threadIdx.x / BACK_SEARCH_COUNT_GPU;
    int ofs = block * THREAD_FACTOR + sub;
    auto control = cont[ofs];

    __shared__ anchor_dt active[THREAD_FACTOR][BACK_SEARCH_COUNT_GPU];
    __shared__ score_dt max_tracker[THREAD_FACTOR][BACK_SEARCH_COUNT_GPU];
    __shared__ parent_dt j_tracker[THREAD_FACTOR][BACK_SEARCH_COUNT_GPU];

    ((short4*)active[sub])[id] = ((short4*)a)[ofs * TILE_SIZE_ACTUAL + id];
    if (control.is_new_read) {
        max_tracker[sub][id] = 0;
        j_tracker[sub][id] = -1;
    } else {
        max_tracker[sub][id] = max_tracker_g[ofs][id];
        j_tracker[sub][id] = j_tracker_g[ofs][id];
    }

#ifdef USE_LOCAL_BUFFER
    __shared__ anchor_dt a_local[THREAD_FACTOR][BACK_SEARCH_COUNT_GPU];
    __shared__ return_dt ret_local[THREAD_FACTOR][BACK_SEARCH_COUNT_GPU];
    ((short4*)a_local[sub])[id] = ((short4*)a)[ofs * TILE_SIZE_ACTUAL + BACK_SEARCH_COUNT_GPU + id];
#endif

    for (int i = BACK_SEARCH_COUNT_GPU, curr_idx = 0; curr_idx < n; i++, curr_idx++) {

        __syncthreads();
        anchor_dt curr;
        *((short4*)&curr) = ((short4*)active[sub])[i % BACK_SEARCH_COUNT_GPU];
        score_dt f_curr = max_tracker[sub][i % BACK_SEARCH_COUNT_GPU];
        parent_dt p_curr = j_tracker[sub][i % BACK_SEARCH_COUNT_GPU];
        if (curr.w >= f_curr) {
            f_curr = curr.w;
            p_curr = (parent_dt)-1;
        }

        /* read in new query anchor, put into active array*/
        __syncthreads();
        if (id == i % BACK_SEARCH_COUNT_GPU) {
#ifdef USE_LOCAL_BUFFER
            active[sub][id] = a_local[sub][id];
#else
            ((short4*)active[sub])[id] =
              ((short4*)a)[ofs * TILE_SIZE_ACTUAL + i];
#endif
            max_tracker[sub][id] = 0;
            j_tracker[sub][id] = -1;
        }

        __syncthreads();
        score_dt sc = chain_dp_score(active[sub], curr,
                control.avg_qspan, max_dist_x, max_dist_y, bw, id);

        __syncthreads();
        if (sc + f_curr >= max_tracker[sub][id]) {
            max_tracker[sub][id] = sc + f_curr;
            j_tracker[sub][id] = (parent_dt)curr_idx +
                (parent_dt)control.tile_num * n;
        }

        __syncthreads();
        if (id == curr_idx % BACK_SEARCH_COUNT_GPU) {
#ifdef USE_LOCAL_BUFFER
            ret_local[sub][id].score = f_curr;
            ret_local[sub][id].parent = p_curr;
#else
            return_dt tmp;
            tmp.score = f_curr;
            tmp.parent = p_curr;
            ((short4*)ret)[ofs * TILE_SIZE + curr_idx] = *((short4*)&tmp);
#endif
        }

#ifdef USE_LOCAL_BUFFER
        if ((i + 1) % BACK_SEARCH_COUNT_GPU == 0) {
            ((short4*)a_local[sub])[id] =
                ((short4*)a)[ofs * TILE_SIZE_ACTUAL + i + 1 + id];
            ((short4*)ret)[ofs * TILE_SIZE +
                    curr_idx - BACK_SEARCH_COUNT_GPU + 1 + id] =
                ((short4*)ret_local[sub])[id];
        }
#endif

    }

    __syncthreads();
    max_tracker_g[ofs][id] = max_tracker[sub][id];
    j_tracker_g[ofs][id] = j_tracker[sub][id];
}
