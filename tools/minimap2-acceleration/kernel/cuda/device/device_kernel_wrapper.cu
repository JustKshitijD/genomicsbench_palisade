#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <ctime>
#include <cstdio>
#include "device_kernel_wrapper.h"
#include "datatypes.h"
#include "common.h"
#include "memory_scheduler.h"


__global__
void device_chain_tiled(
        return_dt *ret, int n, const anchor_dt *a,
        control_dt *control, score_dt **max_tracker, parent_dt **j_tracker,
        int max_dist_x, int max_dist_y, int bw);

__host__
void device_chain_kernel_wrapper(
        std::vector<control_dt> &cont,
        std::vector<anchor_dt> &arg,
        std::vector<return_dt> &ret,
        int max_dist_x, int max_dist_y, int bw)
{
    auto batch_count = cont.size() / PE_NUM;

    control_dt *h_control;
    anchor_dt *h_arg;
    return_dt *h_ret;

    hipHostMalloc(&h_control, cont.size() * sizeof(control_dt));
    hipHostMalloc(&h_arg, arg.size() * sizeof(anchor_dt));
    hipHostMalloc(&h_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
    ret.resize(batch_count * TILE_SIZE * PE_NUM);

    memcpy(h_control, cont.data(), cont.size() * sizeof(control_dt));
    memcpy(h_arg, arg.data(), arg.size() * sizeof(anchor_dt));

    struct timespec start, end;
    clock_gettime(CLOCK_BOOTTIME, &start);

    control_dt *d_control;
    anchor_dt *d_arg;
    return_dt *d_ret;

    // presistent storage
    score_dt *d_max_tracker[PE_NUM];
    parent_dt *d_j_tracker[PE_NUM];

    score_dt **d_d_max_tracker;
    parent_dt **d_d_j_tracker;

    hipMalloc(&d_control, cont.size() * sizeof(control_dt));
    hipMalloc(&d_arg, arg.size() * sizeof(anchor_dt));
    hipMalloc(&d_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
    for (auto pe = 0; pe < PE_NUM; pe++) {
        hipMalloc(&d_max_tracker[pe], BACK_SEARCH_COUNT_GPU * sizeof(score_dt));
        hipMalloc(&d_j_tracker[pe], BACK_SEARCH_COUNT_GPU * sizeof(parent_dt));
    }
    hipMalloc(&d_d_max_tracker, PE_NUM * sizeof(score_dt *));
    hipMalloc(&d_d_j_tracker, PE_NUM * sizeof(parent_dt *));

    hipMemcpy(d_control, h_control,
            cont.size() * sizeof(control_dt), hipMemcpyHostToDevice);
    hipMemcpy(d_arg, h_arg,
            arg.size() * sizeof(anchor_dt), hipMemcpyHostToDevice);
    hipMemcpy(d_d_max_tracker, d_max_tracker,
            PE_NUM * sizeof(score_dt *), hipMemcpyHostToDevice);
    hipMemcpy(d_d_j_tracker, d_j_tracker,
            PE_NUM * sizeof(parent_dt *), hipMemcpyHostToDevice);

    hipStream_t streams[STREAM_NUM];
    for (auto i = 0; i < STREAM_NUM; i++) {
        hipStreamCreate(&streams[i]);
    }

    clock_gettime(CLOCK_BOOTTIME, &end);
    printf(" ***** kernel took %f seconds to transfer in data\n",
        ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / 1E9);

    for (auto batch = 0; batch < batch_count; batch++) {
        for (auto st = 0; st < STREAM_NUM; st++) {
            device_chain_tiled<<<BLOCK_NUM,
                                 THREAD_FACTOR * BACK_SEARCH_COUNT_GPU,
                                 0, streams[st]>>>(
                    d_ret + batch * PE_NUM * TILE_SIZE +
                        st * BLOCK_NUM * THREAD_FACTOR * TILE_SIZE,
                    TILE_SIZE,
                    d_arg + batch * PE_NUM * TILE_SIZE_ACTUAL +
                        st * BLOCK_NUM * THREAD_FACTOR * TILE_SIZE_ACTUAL,
                    d_control + batch * PE_NUM +
                        st * BLOCK_NUM * THREAD_FACTOR,
                    d_d_max_tracker + st * BLOCK_NUM * THREAD_FACTOR,
                    d_d_j_tracker + st * BLOCK_NUM * THREAD_FACTOR,
                    max_dist_x, max_dist_y, bw);
        }
    }

    for (auto i = 0; i < STREAM_NUM; i++) {
        hipStreamSynchronize(streams[i]);
    }

    clock_gettime(CLOCK_BOOTTIME, &end);
    printf(" ***** kernel took %f seconds to transfer in and execute\n",
        ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / 1E9);

    hipMemcpy(h_ret, d_ret,
            batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt),
            hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_BOOTTIME, &end);
    printf(" ***** kernel took %f seconds for end-to-end\n",
        ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / 1E9);

    memcpy(ret.data(), h_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
}

